// -------------------------------
// gpu_matrix_multiplication.cu
// -------------------------------
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16

__global__ void matrixMulKernel(int *A, int *B, int *C) {
    int row = threadIdx.y;
    int col = threadIdx.x;
    int sum = 0;

    for (int k = 0; k < N; ++k) {
        sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
}

int main() {
    int A[N*N], B[N*N], C[N*N];
    int *d_A, *d_B, *d_C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < N*N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    hipMalloc(&d_A, N*N*sizeof(int));
    hipMalloc(&d_B, N*N*sizeof(int));
    hipMalloc(&d_C, N*N*sizeof(int));

    hipMemcpy(d_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);

    hipEventRecord(start);

    matrixMulKernel<<<1, threadsPerBlock>>>(d_A, d_B, d_C);

    hipEventRecord(stop);

    hipMemcpy(C, d_C, N*N*sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Matrix Multiplication completed in %f milliseconds\n", milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
